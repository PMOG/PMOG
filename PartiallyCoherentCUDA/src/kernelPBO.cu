#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuColor.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>

#define MAXTHREADS 512
#define COLORDEPTH 512

uchar4 *d_cmap;
double2 axes=make_double2(4, 4);
double2 origin=make_double2(-2, -2);

inline int ceil(int num, int den){
	return (num+den-1)/den;
}


__host__ __device__ float LaguerreL(float* a, int n, float alpha, float x){
	float temp, yy=0;
	float y=(n>1)?a[n-1]:0;
	for(int k=n-2; k>0; k--){
		temp=y;
		y=a[k]+(2*k+1+alpha-x)/(k+1)*y-(k+1+alpha)/(k+2)*yy;
		yy=temp;
	}
	return a[0]+(1+alpha-x)*y-(1+alpha)*yy/2;
}

__global__ void partiallyCoherent(uchar4* d_pixel, int2 image, uchar4* d_cmap, double2 origin, double2 axes, int2 ensemble){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	if(i<image.x && j<image.y){
		int gid=j*image.y+i;

		// reference frame
		float x=fma((float)i/image.x, (float)axes.x, (float)origin.x);
		float y=fma((float)j/image.y, (float)axes.y, (float)origin.y);

		float I=0;
		float c=0.2;
		float xk, yk, rk;
		for(int k=0; k<ensemble.x; k++){
			xk=x+c*cospi((2*k)/(float)ensemble.x);
			yk=y+c*sinpi((2*k)/(float)ensemble.x);
			rk=sqrtf(xk*xk+yk*yk);
			I+=rk*exp(-rk*rk)*2/ensemble.x;
		}


		int k=(int)(COLORDEPTH*I);
		d_pixel[gid]=d_cmap[clamp(k, 0, COLORDEPTH-1)];
	}
}


void init_kernel(int2 image){
	checkCudaErrors(hipMalloc((void**)&d_cmap, COLORDEPTH*sizeof(uchar4)));
	gray<<<1, COLORDEPTH>>>(d_cmap, COLORDEPTH);
}

void launch_kernel(uchar4* d_pixel, int2 image, float time){
	static const dim3 block(MAXTHREADS);
	static const dim3 grid(ceil(image.x, block.x), ceil(image.y, block.y));
	static const int2 ensemble={16,2};

	partiallyCoherent<<<grid,block>>>(d_pixel, image, d_cmap, origin, axes, ensemble);

	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
}
