#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuColor.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>
#include <hiprand.h>

#define MAXTHREADS 512
#define COLORDEPTH 256

double2 axes=make_double2(4, 4);
double2 origin=make_double2(-2, -2);
int2 ensemble=make_int2(200,200);

uchar4 *d_cmap;
float2 *d_points;

inline int ceil(int num, int den){
	return (num+den-1)/den;
}

__host__ __device__ float LaguerreL(float* a, int n, float alpha, float x){
	float temp, yy=0;
	float y=(n>1)?a[n-1]:0;
	for(int k=n-2; k>0; k--){
		temp=y;
		y=a[k]+(2*k+1+alpha-x)/(k+1)*y-(k+1+alpha)/(k+2)*yy;
		yy=temp;
	}
	return a[0]+(1+alpha-x)*y-(1+alpha)*yy/2;
}

__global__ void diskPointPicking(float2* d_points, int n){
	// maps [0,1]^2 to the unit disk
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n){
		float r=sqrtf(d_points[i].x);
		float sinptr, cosptr;
		sincospif(2*d_points[i].y, &sinptr, &cosptr);
		d_points[i].x=r*cosptr;
		d_points[i].y=r*sinptr;
	}
}


__global__ void blackScreen(uchar4* d_pixel, int2 image){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	if(i<image.x && j<image.y){
		int gid=j*image.y+i;
		d_pixel[gid]=make_uchar4(0x00,0x00,0x00,0xFF);
	}
}

__global__ void partiallyCoherent(uchar4* d_pixel, int2 image, uchar4* d_cmap, float2* d_points, int2 ensemble, double2 axes, double2 origin){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	if(i<image.x && j<image.y){
		int gid=j*image.y+i;

		// reference frame
		float x=fma((float)i/image.x, (float)axes.x, (float)origin.x);
		float y=fma((float)j/image.y, (float)axes.y, (float)origin.y);

		int l=1;
		float I=0.0;
		float c=0.5;
		float xp, yp, rp, tp;
		float u0, sinptr, cosptr;
		float2 uk;
		for(int k=0; k<ensemble.x; k++){
			uk=make_float2(0,0);
			for(int m=0; m<ensemble.y; m++){
				int p=k*ensemble.y+m;
				xp=fma(c, d_points[p].x, x);
				yp=fma(c, d_points[p].y, y);
				rp=hypotf(yp, xp);
				tp=atan2f(yp, xp);

				sincosf(l*tp, &sinptr, &cosptr);
				u0=powf(rp,abs(l))*expf(-rp*rp)*2.35/ensemble.y;
				uk.x=fma(u0, cosptr, uk.x);
				uk.y=fma(u0, sinptr, uk.y);
			}
			I+=(uk.x*uk.x+uk.y*uk.y);
		}

		int cindex=(int)(COLORDEPTH*I/ensemble.x);
		d_pixel[gid]=d_cmap[clamp(cindex, 0, COLORDEPTH-1)];
	}
}


void init_kernel(int2 image){
	hipMalloc((void**)&d_cmap, COLORDEPTH*sizeof(uchar4));
	hot<<<1, COLORDEPTH>>>(d_cmap, COLORDEPTH);


	int npoints=ensemble.x*ensemble.y;
	hipMalloc((void**)&d_points, npoints*sizeof(float2));

	unsigned long long seed=1000;

	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(generator, seed);
	hiprandGenerateUniform(generator, (float*)d_points, 2*npoints);

	const dim3 block(MAXTHREADS);
	const dim3 grid(ceil(npoints, block.x));
	diskPointPicking<<<grid,block>>>(d_points, npoints);
}

void launch_kernel(uchar4* d_pixel, int2 image, float time){
	static const dim3 block(MAXTHREADS);
	static const dim3 grid(ceil(image.x, block.x), ceil(image.y, block.y));
	static int count=0;
	if(count<60){
		blackScreen<<<grid,block>>>(d_pixel, image);
	}else if(count==60){
		partiallyCoherent<<<grid,block>>>(d_pixel, image, d_cmap, d_points, ensemble, axes, origin);
	}
	count++;
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
}
