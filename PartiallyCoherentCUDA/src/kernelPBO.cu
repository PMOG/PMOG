#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuColor.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>
#include <hiprand.h>

#define MAXTHREADS 512
#define COLORDEPTH 256
#define WEIGHT 0.07

double2 axes=make_double2(4, 4);
double2 origin=make_double2(-2, -2);
int2 ensemble=make_int2(200,200);

uchar4 *d_cmap;
float2 *d_points;
float2 *d_field;
float *d_intensity;
float *d_chi;

inline int ceil(int num, int den){
	return (num+den-1)/den;
}

__host__ __device__ float LaguerreL(float* a, int n, float alpha, float x){
	float temp, yy=0;
	float y=(n>1)?a[n-1]:0;
	for(int k=n-2; k>0; k--){
		temp=y;
		y=a[k]+(2*k+1+alpha-x)/(k+1)*y-(k+1+alpha)/(k+2)*yy;
		yy=temp;
	}
	return a[0]+(1+alpha-x)*y-(1+alpha)*yy/2;
}

__global__ void diskPointPicking(float2* d_points, int n){
	// maps [0,1]^2 to the unit disk
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n){
		float r=sqrtf(d_points[i].x);
		float sinptr, cosptr;
		sincospif(2*d_points[i].y, &sinptr, &cosptr);
		d_points[i].x=r*cosptr;
		d_points[i].y=r*sinptr;
	}
}

__global__ void coherentSum(uchar4 *d_pixel, int2 image, uchar4* d_cmap, float2 *d_points, float2 *d_field, float *d_intensity, int k, int2 ensemble, double2 axes, double2 origin){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	if(i<image.x && j<image.y){
		int gid=j*image.x+i;

		// reference frame
		float x=fma((float)i/image.x, (float)axes.x, (float)origin.x);
		float y=fma((float)j/image.y, (float)axes.y, (float)origin.y);

		int l=1;
		float c=1;
		float xp, yp, rp, tp, u0, sinptr, cosptr;
		float2 uk=make_float2(0,0);
		for(int m=0; m<ensemble.y; m++){
			int p=k*ensemble.y+m;
			xp=fma(c, d_points[p].x, x);
			yp=fma(c, d_points[p].y, y);
			rp=hypotf(yp, xp);
			tp=atan2f(yp, xp);

			sincosf(l*tp, &sinptr, &cosptr);
			u0=powf(rp,abs(l))*expf(-rp*rp);
			uk.x=fma(u0, cosptr, uk.x);
			uk.y=fma(u0, sinptr, uk.y);
		}
		float uu=uk.x*uk.x+uk.y*uk.y;
		d_field[gid]=uk;
		d_intensity[gid]+=uu;
		int cindex=(int)(COLORDEPTH*WEIGHT*uu/ensemble.y);
		d_pixel[gid]=d_cmap[clamp(cindex, 0, COLORDEPTH-1)];
	}
}

__global__ void crossCorrelation(float* d_chi, float2* d_field, int2 image){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	if(i<image.x && j<image.y){
		int gid=j*image.x+i;
		int rid=(image.y-1-j)*image.x+(image.x-1-i); // rotated index

		d_chi[gid]+=d_field[gid].x*d_field[rid].x+d_field[gid].y*d_field[rid].y;
	}
}

__global__ void average(uchar4* d_pixel, int2 image, uchar4* d_cmap, float *d_intensity, int2 ensemble){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	if(i<image.x && j<image.y){
		int gid=j*image.x+i;
		float average=d_intensity[gid]/(ensemble.x*ensemble.y);
		int cindex=(int)(COLORDEPTH*average);
		d_pixel[gid]=d_cmap[clamp(cindex, 0, COLORDEPTH-1)];
	}
}

void init_kernel(int2 image){
	// Initialize colormap
	hipMalloc((void**)&d_cmap, COLORDEPTH*sizeof(uchar4));
	hot<<<1, COLORDEPTH>>>(d_cmap, COLORDEPTH);

	// Allocate field, intensity, and cross-correlation
	int npixels=image.x*image.y;
	hipMalloc((void**)&d_field, npixels*sizeof(float2));
	hipMalloc((void**)&d_intensity, npixels*sizeof(float));
	hipMemset(d_intensity, 0, npixels*sizeof(float));
	hipMalloc((void**)&d_chi, npixels*sizeof(float));
	hipMemset(d_chi, 0, npixels*sizeof(float));

	// Allocate points
	int npoints=ensemble.x*ensemble.y;
	hipMalloc((void**)&d_points, npoints*sizeof(float2));

	// Generate random distribution
	unsigned long long seed=1000;
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(generator, seed);
	hiprandGenerateUniform(generator, (float*)d_points, 2*npoints);

	// Map to unit disk
	const dim3 block(MAXTHREADS);
	const dim3 grid(ceil(npoints, block.x));
	diskPointPicking<<<grid,block>>>(d_points, npoints);
}

void launch_kernel(uchar4* d_pixel, int2 image, float time){
	static const dim3 block(MAXTHREADS);
	static const dim3 grid(ceil(image.x, block.x), ceil(image.y, block.y));
	static int k=0;

	if(k<ensemble.x){
		coherentSum<<<grid,block>>>(d_pixel, image, d_cmap, d_points, d_field, d_intensity, k, ensemble, axes, origin);
		crossCorrelation<<<grid,block>>>(d_chi, d_field, image);
	}else{
		average<<<grid,block>>>(d_pixel, image, d_cmap, d_chi, ensemble);
	}
	k++;
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
}
