#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuColor.h"
#include <hip/hip_runtime_api.h>
#include <hiprand.h>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <float.h>

#define NSTREAMS 1
#define MAXTHREADS 512
#define COLORDEPTH 256

double2 axes=make_double2(4, 4);
double2 origin=make_double2(-2, -2);
int2 ensemble=make_int2(200,200);

uchar4 *d_cmap;
float3 *d_points;
float2 *d_field;
float *d_intensity, *d_chi;


int npixels;
hipStream_t stream[NSTREAMS];



inline int ceil(int num, int den){
	return (num+den-1)/den;
}

__host__ __device__ float LaguerreL(float* a, int n, float alpha, float x){
	float temp, yy=0;
	float y=(n>1)?a[n-1]:0;
	for(int k=n-2; k>0; k--){
		temp=y;
		y=a[k]+(2*k+1+alpha-x)/(k+1)*y-(k+1+alpha)/(k+2)*yy;
		yy=temp;
	}
	return a[0]+(1+alpha-x)*y-(1+alpha)*yy/2;
}

__global__ void diskPointPicking(int n, float3* d_points){
	// maps [0,1]^2 to the unit disk
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n){
		float r=sqrtf(d_points[i].x);
		float sinptr, cosptr;
		sincospif(2*d_points[i].y, &sinptr, &cosptr);
		d_points[i].x=r*cosptr;
		d_points[i].y=r*sinptr;
	}
}

__global__ void ensembleMember(int2 image, float2 *d_field, float *d_intensity, float3 *d_points, int k, int2 ensemble, double2 axes, double2 origin){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	if(i<image.x && j<image.y){
		int gid=j*image.x+i;

		// reference frame
		float x=fma((float)i/image.x, (float)axes.x, (float)origin.x);
		float y=fma((float)j/image.y, (float)axes.y, (float)origin.y);

		int l=3;
		float c=0.7;
		float xp, yp, rp, tp, u0, sinptr, cosptr;
		float2 uk=make_float2(0,0);
		for(int m=0; m<ensemble.y; m++){
			int p=k*ensemble.y+m;
			xp=fma(c, d_points[p].x, x);
			yp=fma(c, d_points[p].y, y);
			rp=hypotf(yp, xp);
			tp=atan2f(yp, xp);

			sincosf(l*tp+2*M_PI*d_points[p].z, &sinptr, &cosptr);
			u0=powf(rp,abs(l))*expf(-rp*rp);
			uk.x=fma(u0, cosptr, uk.x);
			uk.y=fma(u0, sinptr, uk.y);
		}
		float uu=uk.x*uk.x+uk.y*uk.y;
		d_field[gid]=uk;
		atomicAdd(&d_intensity[gid], uu);
	}
}

__global__ void crossCorrelation(int2 image, float* d_chi, float2* d_field){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	if(i<image.x && j<image.y){
		int gid=j*image.x+i;
		int rid=(image.y-1-j)*image.x+(image.x-1-i); // rotated index
		atomicAdd(&d_chi[gid], d_field[gid].x*d_field[rid].x+d_field[gid].y*d_field[rid].y);
	}
}

__global__ void imagesc(int2 image, uchar4* d_pixel, uchar4* d_cmap, float *d_X, float xmin, float xmax){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	if(i<image.x && j<image.y){
		int gid=j*image.x+i;
		int cid=clamp((int)((COLORDEPTH-1)*(d_X[gid]-xmin)/(xmax-xmin)), 0, COLORDEPTH-1);
		d_pixel[gid]=d_cmap[cid];
	}
}


template<typename T> T thrustMax(int n, T* d_x){
	thrust::device_ptr<T> t_x(d_x);
	return thrust::reduce(t_x, t_x+n, -FLT_MAX, thrust::maximum<T>());
}


template<typename T> T thrustMin(int n, T* d_x){
	thrust::device_ptr<T> t_x(d_x);
	return thrust::reduce(t_x, t_x+n,  FLT_MAX, thrust::minimum<T>());
}


void init_kernel(int2 image){
	npixels=image.x*image.y;

	// Stream creation
	for(int i=0; i<NSTREAMS; i++){
		hipStreamCreate(&stream[i]);
	}

	// Initialize colormap
	hipMalloc((void**)&d_cmap, COLORDEPTH*sizeof(uchar4));
	gray<<<1, COLORDEPTH>>>(d_cmap, COLORDEPTH);

	// Allocate field, intensity, and cross-correlation
	hipMalloc((void**)&d_field, NSTREAMS*npixels*sizeof(float2));
	hipMalloc((void**)&d_intensity, npixels*sizeof(float));
	hipMemset(d_intensity, 0, npixels*sizeof(float));
	hipMalloc((void**)&d_chi, npixels*sizeof(float));
	hipMemset(d_chi, 0, npixels*sizeof(float));

	// Allocate points
	int npoints=ensemble.x*ensemble.y;
	hipMalloc((void**)&d_points, npoints*sizeof(float3));

	// Generate random distribution
	unsigned long long seed=1000;
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(generator, seed);
	hiprandGenerateUniform(generator, (float*)d_points, 3*npoints);

	// Map to unit disk
	const dim3 block(MAXTHREADS);
	const dim3 grid(ceil(npoints, block.x));
	diskPointPicking<<<grid,block>>>(npoints, d_points);
}

void launch_kernel(int2 image, uchar4* d_pixel, float time){
	static const dim3 block(MAXTHREADS);
	static const dim3 grid(ceil(image.x, block.x), ceil(image.y, block.y));
	static int k=0;

	if(k<ensemble.x){
		for(int i=0; i<NSTREAMS; i++){
			int offset=i*npixels;
			ensembleMember<<<grid,block,0,stream[i]>>>(image, d_field+offset, d_intensity, d_points, k, ensemble, axes, origin);
			crossCorrelation<<<grid,block,0,stream[i]>>>(image, d_chi, d_field+offset);
		}
		k+=NSTREAMS;
	}

	float min = thrustMin(npixels, d_intensity);
	float max = thrustMax(npixels, d_intensity);
	imagesc<<<grid,block>>>(image, d_pixel, d_cmap, d_intensity, min, max);

	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
}
